#include "hip/hip_runtime.h"
#ifndef SINDIVIDUAL_CU
#define SINDIVIDUAL_CU

#include <hiprand/hiprand_kernel.h>
#include "individual.cu"

template<typename DNA, typename Tfitness, int Size>
class SIndividual : public Individual<DNA, Tfitness, Size>{
public:
    __device__ void random(int tid, DNA a, DNA b);
};

template<typename DNA, typename Tfitness, int Size>
__device__ void SIndividual<DNA, Tfitness, Size>::random(int tid, DNA a, DNA b){
    //printf("xd");
    hiprandState state;
    hiprand_init(clock64(), tid, 0, &state);
    for(int i = 0; i < Size; i++){
        this->genes[i] = generateRandomNumber(&state, a, b);
    }
}

#endif // SINDIVIDUAL_CU